#include "cu_common.h"

namespace blas { namespace cuda {

cuShape_t::cuShape_t(const shape_t& shape) : ndims(shape.size()) {
    CU_CHECK_ERR(hipMalloc(&sizes, ndims));
    CU_CHECK_ERR(hipMemcpy(sizes, shape.data(), ndims * sizeof(size_t),
                            hipMemcpyKind::hipMemcpyHostToDevice));
}

cuShape_t::~cuShape_t() { hipFree(sizes); }

cuIndex_t::cuIndex_t(const index_t& shape) : ndims(shape.size()) {
    CU_CHECK_ERR(hipMalloc(&items, ndims));
    CU_CHECK_ERR(hipMemcpy(items, shape.data(), ndims * sizeof(size_t),
                            hipMemcpyKind::hipMemcpyHostToDevice));
}

cuIndex_t::~cuIndex_t() { hipFree(items); }
}  // namespace cuda
}  // namespace blas